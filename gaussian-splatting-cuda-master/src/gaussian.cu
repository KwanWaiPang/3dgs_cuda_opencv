#include "hip/hip_runtime.h"
#include "debug_utils.cuh"
#include "gaussian.cuh"
#include "read_utils.cuh"
#include <exception>
#include <thread>

GaussianModel::GaussianModel(int sh_degree) : _max_sh_degree(sh_degree) {
}

torch::Tensor GaussianModel::Get_covariance(float scaling_modifier) {
    auto L = build_scaling_rotation(scaling_modifier * Get_scaling(), _rotation);
    auto actual_covariance = torch::mm(L, L.transpose(1, 2));
    auto symm = strip_symmetric(actual_covariance);
    return symm;
}

/**
 * @brief Fetches the features of the Gaussian model
 *
 * This function concatenates _features_dc and _features_rest along the second dimension.
 *
 * @return Tensor of the concatenated features
 */
torch::Tensor GaussianModel::Get_features() const {
    auto features_dc = _features_dc;
    auto features_rest = _features_rest;
    return torch::cat({features_dc, features_rest}, 1);
}

/**
 * @brief Increment the SH degree by 1
 *
 * This function increments the active_sh_degree by 1, up to a maximum of max_sh_degree.
 */
void GaussianModel::One_up_sh_degree() {
    if (_active_sh_degree < _max_sh_degree) {
        _active_sh_degree++;
    }
}

/**
 * @brief Initialize Gaussian Model from a Point Cloud.
 *
 * This function creates a Gaussian model from a given PointCloud object. It also sets
 * the spatial learning rate scale. The model's features, scales, rotations, and opacities
 * are initialized based on the input point cloud.
 *
 * @param pcd The input point cloud
 * @param spatial_lr_scale The spatial learning rate scale
 */
void GaussianModel::Create_from_pcd(PointCloud& pcd, float spatial_lr_scale) {
    _spatial_lr_scale = spatial_lr_scale;//相机空间的学习率（相机分布的空间）

    const auto pointType = torch::TensorOptions().dtype(torch::kFloat32);
    //从点云数据 pcd 中创建了一个大小为点数×3的张量 _xyz，用于存储点的坐标。张量类型为 torch::kCUDA，表示在 CUDA 上进行计算，并设置为需要梯度计算。
    _xyz = torch::from_blob(pcd._points.data(), {static_cast<long>(pcd._points.size()), 3}, pointType).to(torch::kCUDA).set_requires_grad(true);
    
    // 计算了点云中每个点到原点的欧式距离的平方，并进行了最小值截断，避免出现零距离
    //  distCUDA2 的函数，用于计算点集中每个点到其余所有点的欧式距离的平均值
    auto dist2 = torch::clamp_min(distCUDA2(_xyz), 0.0000001);//计算点云中每个点的距离平方，并进行最小值截断以确保不会出现零距离。
    
    // 计算了每个点的缩放系数，并进行了对数变换。然后将其扩展为与点的坐标相同的形状，并设置为需要梯度计算。
    _scaling = torch::log(torch::sqrt(dist2)).unsqueeze(-1).repeat({1, 3}).to(torch::kCUDA, true).set_requires_grad(true);
    
    // 创建了一个大小为点数×4的张量 _rotation，用于存储旋转参数，初始化为单位四元数
    _rotation = torch::zeros({_xyz.size(0), 4}).index_put_({torch::indexing::Slice(), 0}, 1).to(torch::kCUDA, true).set_requires_grad(true);
    
    // 用于存储每个点的不透明度参数，通过逆 Sigmoid 函数初始化。然后将其扩展为与点的坐标相同的形状，并设置为需要梯度计算。
    _opacity = inverse_sigmoid(0.5 * torch::ones({_xyz.size(0), 1})).to(torch::kCUDA, true).set_requires_grad(true);
    
    // 用于存储每个点的最大半径
    _max_radii2D = torch::zeros({_xyz.size(0)}).to(torch::kCUDA, true);

    // colors
    auto colorType = torch::TensorOptions().dtype(torch::kUInt8);//用于指定颜色数据的数据类型为无符号 8 位整数
    // 将点云的颜色数据转换为球谐系数，并将其存储在 fused_color 张量中。
    auto fused_color = RGB2SH(torch::from_blob(pcd._colors.data(), {static_cast<long>(pcd._colors.size()), 3}, colorType).to(pointType) / 255.f).to(torch::kCUDA);

    // features
    //features存储了点云的颜色数据，其形状为点数×3×(max_sh_degree+1)^2
    auto features = torch::zeros({fused_color.size(0), 3, static_cast<long>(std::pow((_max_sh_degree + 1), 2))}).to(torch::kCUDA);
    //将颜色信息复制到 features 张量的前三个通道中。
    features.index_put_({torch::indexing::Slice(), torch::indexing::Slice(torch::indexing::None, 3), 0}, fused_color);
    //将 features 张量的后面通道中的值初始化为0。
    features.index_put_({torch::indexing::Slice(), torch::indexing::Slice(3, torch::indexing::None), torch::indexing::Slice(1, torch::indexing::None)}, 0.0);
    
    //从 features 张量中提取了前一个通道的数据，并将其转置为形状为 (fused_color.size(0), 1, 3) 的张量 _features_dc，并设置为需要计算梯度。
    _features_dc = features.index({torch::indexing::Slice(), torch::indexing::Slice(), torch::indexing::Slice(0, 1)}).transpose(1, 2).contiguous().set_requires_grad(true);
    //从 features 张量中提取了除前一个通道以外的数据，并将其转置为形状为 (fused_color.size(0), (_max_sh_degree + 1)^2 - 1, 3) 的张量 _features_rest，并设置为需要计算梯度。
    _features_rest = features.index({torch::indexing::Slice(), torch::indexing::Slice(), torch::indexing::Slice(1, torch::indexing::None)}).transpose(1, 2).contiguous().set_requires_grad(true);

    //输出点云的数量
    std::cout << "Number of points at initialisation of Gaussian Function: " << _xyz.size(0) << std::endl;
}

/**
 * @brief Setup the Gaussian Model for training （用于设置模型训练的参数和优化器。根据给定的优化参数初始化模型的优化器，并设置学习率、参数组和其他优化器参数。）
 *
 * This function sets up the Gaussian model for training by initializing several
 * parameters and settings based on the provided OptimizationParameters object.
 *
 * @param params The OptimizationParameters object providing the settings for training
 */
void GaussianModel::Training_setup(const gs::param::OptimizationParameters& params) {
    this->_percent_dense = params.percent_dense;//设置了密集度参数
    this->_xyz_gradient_accum = torch::zeros({this->_xyz.size(0), 1}).to(torch::kCUDA);//创建了一个与点集大小相同的零张量，用于存储梯度累积，并转移到CUDA设备上
    this->_denom = torch::zeros({this->_xyz.size(0), 1}).to(torch::kCUDA);//创建了一个与点集大小相同的零张量 _denom，用于计算优化步骤中的分母，并将其移到 CUDA 设备上。
    this->_xyz_scheduler_args = Expon_lr_func(params.position_lr_init * this->_spatial_lr_scale,
                                              params.position_lr_final * this->_spatial_lr_scale,
                                              params.position_lr_delay_mult,
                                              params.position_lr_max_steps);//根据给定参数初始化指数衰减的学习率对象，在优化过程中用于调整点集参数的学习率。

    std::vector<torch::optim::OptimizerParamGroup> optimizer_params_groups;//创建了一个优化器参数组的空向量。
    optimizer_params_groups.reserve(6);//优化器参数组的大小设置为6
    // 向优化器参数组中添加了一个参数组，该参数组包含了点集参数 _xyz 和相应的 Adam 优化器参数。
    optimizer_params_groups.push_back(torch::optim::OptimizerParamGroup({_xyz}, std::make_unique<torch::optim::AdamOptions>(params.position_lr_init * this->_spatial_lr_scale)));
    //添加颜色特征参数
    optimizer_params_groups.push_back(torch::optim::OptimizerParamGroup({_features_dc}, std::make_unique<torch::optim::AdamOptions>(params.feature_lr)));
    optimizer_params_groups.push_back(torch::optim::OptimizerParamGroup({_features_rest}, std::make_unique<torch::optim::AdamOptions>(params.feature_lr / 20.)));
    //添加缩放参数
    optimizer_params_groups.push_back(torch::optim::OptimizerParamGroup({_scaling}, std::make_unique<torch::optim::AdamOptions>(params.scaling_lr * this->_spatial_lr_scale)));
    //添加旋转参数
    optimizer_params_groups.push_back(torch::optim::OptimizerParamGroup({_rotation}, std::make_unique<torch::optim::AdamOptions>(params.rotation_lr)));
    //添加不透明度参数
    optimizer_params_groups.push_back(torch::optim::OptimizerParamGroup({_opacity}, std::make_unique<torch::optim::AdamOptions>(params.opacity_lr)));
    
    //设置了每个参数组中 Adam 优化器的 epsilon 值，用于数值稳定性。
    static_cast<torch::optim::AdamOptions&>(optimizer_params_groups[0].options()).eps(1e-15);
    static_cast<torch::optim::AdamOptions&>(optimizer_params_groups[1].options()).eps(1e-15);
    static_cast<torch::optim::AdamOptions&>(optimizer_params_groups[2].options()).eps(1e-15);
    static_cast<torch::optim::AdamOptions&>(optimizer_params_groups[3].options()).eps(1e-15);
    static_cast<torch::optim::AdamOptions&>(optimizer_params_groups[4].options()).eps(1e-15);
    static_cast<torch::optim::AdamOptions&>(optimizer_params_groups[5].options()).eps(1e-15);

    //创建了一个 Adam 优化器对象 _optimizer，并使用之前定义的参数组进行初始化。
    _optimizer = std::make_unique<torch::optim::Adam>(optimizer_params_groups, torch::optim::AdamOptions(0.f).eps(1e-15));
}

void GaussianModel::Update_learning_rate(float iteration) {
    // This is hacky because you cant change in libtorch individual parameter learning rate
    // xyz is added first, since _optimizer->param_groups() return a vector, we assume that xyz stays first
    auto lr = _xyz_scheduler_args(iteration);
    static_cast<torch::optim::AdamOptions&>(_optimizer->param_groups()[0].options()).set_lr(lr);
}

void GaussianModel::Reset_opacity() {
    // opacitiy activation
    auto new_opacity = inverse_sigmoid(torch::ones_like(_opacity, torch::TensorOptions().dtype(torch::kFloat32)) * 0.01f);

    auto adamParamStates = std::make_unique<torch::optim::AdamParamState>(static_cast<torch::optim::AdamParamState&>(
        *_optimizer->state()[c10::guts::to_string(_optimizer->param_groups()[5].params()[0].unsafeGetTensorImpl())]));

    _optimizer->state().erase(c10::guts::to_string(_optimizer->param_groups()[5].params()[0].unsafeGetTensorImpl()));

    adamParamStates->exp_avg(torch::zeros_like(new_opacity));
    adamParamStates->exp_avg_sq(torch::zeros_like(new_opacity));
    // replace tensor
    _optimizer->param_groups()[5].params()[0] = new_opacity.set_requires_grad(true);
    _opacity = _optimizer->param_groups()[5].params()[0];

    _optimizer->state()[c10::guts::to_string(_optimizer->param_groups()[5].params()[0].unsafeGetTensorImpl())] = std::move(adamParamStates);
}

void prune_optimizer(torch::optim::Adam* optimizer, const torch::Tensor& mask, torch::Tensor& old_tensor, int param_position) {
    auto adamParamStates = std::make_unique<torch::optim::AdamParamState>(static_cast<torch::optim::AdamParamState&>(
        *optimizer->state()[c10::guts::to_string(optimizer->param_groups()[param_position].params()[0].unsafeGetTensorImpl())]));
    optimizer->state().erase(c10::guts::to_string(optimizer->param_groups()[param_position].params()[0].unsafeGetTensorImpl()));

    adamParamStates->exp_avg(adamParamStates->exp_avg().index_select(0, mask));
    adamParamStates->exp_avg_sq(adamParamStates->exp_avg_sq().index_select(0, mask));

    optimizer->param_groups()[param_position].params()[0] = old_tensor.index_select(0, mask).set_requires_grad(true);
    old_tensor = optimizer->param_groups()[param_position].params()[0]; // update old tensor
    optimizer->state()[c10::guts::to_string(optimizer->param_groups()[param_position].params()[0].unsafeGetTensorImpl())] = std::move(adamParamStates);
}

void GaussianModel::prune_points(torch::Tensor mask) {
    // reverse to keep points
    auto valid_point_mask = ~mask;
    int true_count = valid_point_mask.sum().item<int>();
    auto indices = torch::nonzero(valid_point_mask == true).index({torch::indexing::Slice(torch::indexing::None, torch::indexing::None), torch::indexing::Slice(torch::indexing::None, 1)}).squeeze(-1);
    prune_optimizer(_optimizer.get(), indices, _xyz, 0);
    prune_optimizer(_optimizer.get(), indices, _features_dc, 1);
    prune_optimizer(_optimizer.get(), indices, _features_rest, 2);
    prune_optimizer(_optimizer.get(), indices, _scaling, 3);
    prune_optimizer(_optimizer.get(), indices, _rotation, 4);
    prune_optimizer(_optimizer.get(), indices, _opacity, 5);

    _xyz_gradient_accum = _xyz_gradient_accum.index_select(0, indices);
    _denom = _denom.index_select(0, indices);
    _max_radii2D = _max_radii2D.index_select(0, indices);
}

void cat_tensors_to_optimizer(torch::optim::Adam* optimizer,
                              torch::Tensor& extension_tensor,
                              torch::Tensor& old_tensor,
                              int param_position) {
    auto adamParamStates = std::make_unique<torch::optim::AdamParamState>(static_cast<torch::optim::AdamParamState&>(
        *optimizer->state()[c10::guts::to_string(optimizer->param_groups()[param_position].params()[0].unsafeGetTensorImpl())]));
    optimizer->state().erase(c10::guts::to_string(optimizer->param_groups()[param_position].params()[0].unsafeGetTensorImpl()));

    adamParamStates->exp_avg(torch::cat({adamParamStates->exp_avg(), torch::zeros_like(extension_tensor)}, 0));
    adamParamStates->exp_avg_sq(torch::cat({adamParamStates->exp_avg_sq(), torch::zeros_like(extension_tensor)}, 0));

    optimizer->param_groups()[param_position].params()[0] = torch::cat({old_tensor, extension_tensor}, 0).set_requires_grad(true);
    old_tensor = optimizer->param_groups()[param_position].params()[0];

    optimizer->state()[c10::guts::to_string(optimizer->param_groups()[param_position].params()[0].unsafeGetTensorImpl())] = std::move(adamParamStates);
}

void GaussianModel::densification_postfix(torch::Tensor& new_xyz,
                                          torch::Tensor& new_features_dc,
                                          torch::Tensor& new_features_rest,
                                          torch::Tensor& new_scaling,
                                          torch::Tensor& new_rotation,
                                          torch::Tensor& new_opacity) {
    cat_tensors_to_optimizer(_optimizer.get(), new_xyz, _xyz, 0);
    cat_tensors_to_optimizer(_optimizer.get(), new_features_dc, _features_dc, 1);
    cat_tensors_to_optimizer(_optimizer.get(), new_features_rest, _features_rest, 2);
    cat_tensors_to_optimizer(_optimizer.get(), new_scaling, _scaling, 3);
    cat_tensors_to_optimizer(_optimizer.get(), new_rotation, _rotation, 4);
    cat_tensors_to_optimizer(_optimizer.get(), new_opacity, _opacity, 5);

    _xyz_gradient_accum = torch::zeros({_xyz.size(0), 1}).to(torch::kCUDA);
    _denom = torch::zeros({_xyz.size(0), 1}).to(torch::kCUDA);
    _max_radii2D = torch::zeros({_xyz.size(0)}).to(torch::kCUDA);
}

void GaussianModel::densify_and_split(torch::Tensor& grads, float grad_threshold, float scene_extent, float min_opacity, float max_screen_size) {
    static const int N = 2;
    const int n_init_points = _xyz.size(0);
    // Extract points that satisfy the gradient condition
    torch::Tensor padded_grad = torch::zeros({n_init_points}).to(torch::kCUDA);
    padded_grad.slice(0, 0, grads.size(0)) = grads.squeeze();
    torch::Tensor selected_pts_mask = torch::where(padded_grad >= grad_threshold, torch::ones_like(padded_grad).to(torch::kBool), torch::zeros_like(padded_grad).to(torch::kBool));
    selected_pts_mask = torch::logical_and(selected_pts_mask, std::get<0>(Get_scaling().max(1)) > _percent_dense * scene_extent);
    auto indices = torch::nonzero(selected_pts_mask.squeeze(-1) == true).index({torch::indexing::Slice(torch::indexing::None, torch::indexing::None), torch::indexing::Slice(torch::indexing::None, 1)}).squeeze(-1);

    torch::Tensor stds = Get_scaling().index_select(0, indices).repeat({N, 1});
    torch::Tensor means = torch::zeros({stds.size(0), 3}).to(torch::kCUDA);
    torch::Tensor samples = torch::randn({stds.size(0), stds.size(1)}).to(torch::kCUDA) * stds + means;
    torch::Tensor rots = build_rotation(_rotation.index_select(0, indices)).repeat({N, 1, 1});

    torch::Tensor new_xyz = torch::bmm(rots, samples.unsqueeze(-1)).squeeze(-1) + _xyz.index_select(0, indices).repeat({N, 1});
    torch::Tensor new_scaling = torch::log(Get_scaling().index_select(0, indices).repeat({N, 1}) / (0.8 * N));
    torch::Tensor new_rotation = _rotation.index_select(0, indices).repeat({N, 1});
    torch::Tensor new_features_dc = _features_dc.index_select(0, indices).repeat({N, 1, 1});
    torch::Tensor new_features_rest = _features_rest.index_select(0, indices).repeat({N, 1, 1});
    torch::Tensor new_opacity = _opacity.index_select(0, indices).repeat({N, 1});

    densification_postfix(new_xyz, new_features_dc, new_features_rest, new_scaling, new_rotation, new_opacity);

    torch::Tensor prune_filter = torch::cat({selected_pts_mask.squeeze(-1), torch::zeros({N * selected_pts_mask.sum().item<int>()}).to(torch::kBool).to(torch::kCUDA)});
    // torch::Tensor prune_filter = torch::cat({selected_pts_mask.squeeze(-1), torch::zeros({N * selected_pts_mask.sum().item<int>()})}).to(torch::kBool).to(torch::kCUDA);
    prune_filter = torch::logical_or(prune_filter, (Get_opacity() < min_opacity).squeeze(-1));
    prune_points(prune_filter);
}

void GaussianModel::densify_and_clone(torch::Tensor& grads, float grad_threshold, float scene_extent) {
    // Extract points that satisfy the gradient condition
    torch::Tensor selected_pts_mask = torch::where(torch::linalg::vector_norm(grads, {2}, 1, true, torch::kFloat32) >= grad_threshold,
                                                   torch::ones_like(grads.index({torch::indexing::Slice()})).to(torch::kBool),
                                                   torch::zeros_like(grads.index({torch::indexing::Slice()})).to(torch::kBool))
                                          .to(torch::kLong);

    selected_pts_mask = torch::logical_and(selected_pts_mask, std::get<0>(Get_scaling().max(1)).unsqueeze(-1) <= _percent_dense * scene_extent);

    auto indices = torch::nonzero(selected_pts_mask.squeeze(-1) == true).index({torch::indexing::Slice(torch::indexing::None, torch::indexing::None), torch::indexing::Slice(torch::indexing::None, 1)}).squeeze(-1);
    torch::Tensor new_xyz = _xyz.index_select(0, indices);
    torch::Tensor new_features_dc = _features_dc.index_select(0, indices);
    torch::Tensor new_features_rest = _features_rest.index_select(0, indices);
    torch::Tensor new_opacity = _opacity.index_select(0, indices);
    torch::Tensor new_scaling = _scaling.index_select(0, indices);
    torch::Tensor new_rotation = _rotation.index_select(0, indices);

    densification_postfix(new_xyz, new_features_dc, new_features_rest, new_scaling, new_rotation, new_opacity);
}

void GaussianModel::Densify_and_prune(float max_grad, float min_opacity, float extent, float max_screen_size) {
    torch::Tensor grads = _xyz_gradient_accum / _denom;
    grads.index_put_({grads.isnan()}, 0.0);

    densify_and_clone(grads, max_grad, extent);
    densify_and_split(grads, max_grad, extent, min_opacity, max_screen_size);
}

void GaussianModel::Add_densification_stats(torch::Tensor& viewspace_point_tensor, torch::Tensor& update_filter) {
    _xyz_gradient_accum.index_put_({update_filter}, _xyz_gradient_accum.index_select(0, update_filter.nonzero().squeeze()) + viewspace_point_tensor.grad().index_select(0, update_filter.nonzero().squeeze()).slice(1, 0, 2).norm(2, -1, true));
    _denom.index_put_({update_filter}, _denom.index_select(0, update_filter.nonzero().squeeze()) + 1);
}

std::vector<std::string> GaussianModel::construct_list_of_attributes() {
    std::vector<std::string> attributes = {"x", "y", "z", "nx", "ny", "nz"};

    for (int i = 0; i < _features_dc.size(1) * _features_dc.size(2); ++i)
        attributes.push_back("f_dc_" + std::to_string(i));

    for (int i = 0; i < _features_rest.size(1) * _features_rest.size(2); ++i)
        attributes.push_back("f_rest_" + std::to_string(i));

    attributes.emplace_back("opacity");

    for (int i = 0; i < _scaling.size(1); ++i)
        attributes.push_back("scale_" + std::to_string(i));

    for (int i = 0; i < _rotation.size(1); ++i)
        attributes.push_back("rot_" + std::to_string(i));

    return attributes;
}

void GaussianModel::Save_ply(const std::filesystem::path& file_path, int iteration, bool isLastIteration) {
    std::cout << "Saving at " << std::to_string(iteration) << " iterations\n";
    auto folder = file_path / ("point_cloud/iteration_" + std::to_string(iteration));
    std::filesystem::create_directories(folder);

    auto xyz = _xyz.cpu().contiguous();
    auto normals = torch::zeros_like(xyz);
    auto f_dc = _features_dc.transpose(1, 2).flatten(1).cpu().contiguous();
    auto f_rest = _features_rest.transpose(1, 2).flatten(1).cpu().contiguous();
    auto opacities = _opacity.cpu();
    auto scale = _scaling.cpu();
    auto rotation = _rotation.cpu();

    std::vector<torch::Tensor> tensor_attributes = {xyz.clone(),
                                                    normals.clone(),
                                                    f_dc.clone(),
                                                    f_rest.clone(),
                                                    opacities.clone(),
                                                    scale.clone(),
                                                    rotation.clone()};
    auto attributes = construct_list_of_attributes();
    std::thread t = std::thread([folder, tensor_attributes, attributes]() {
        Write_output_ply(folder / "point_cloud.ply", tensor_attributes, attributes);
    });

    if (isLastIteration) {
        t.join();
    } else {
        t.detach();
    }
}