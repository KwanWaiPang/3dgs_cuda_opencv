#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "hip/hip_runtime.h"
#include ""
#include "rasterizer_impl.h"
#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <numeric>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "backward.h"
#include "forward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n) {
    uint32_t msb = sizeof(n) * 4;
    uint32_t step = msb;
    while (step > 1) {
        step /= 2;
        if (n >> msb)
            msb += step;
        else
            msb -= step;
    }
    if (n >> msb)
        msb++;
    return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
                             const float* orig_points,
                             const float* viewmatrix,
                             const float* projmatrix,
                             bool* present) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= P)
        return;

    float3 p_view;
    present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps.
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
    int P,
    const float2* points_xy,
    const float* depths,
    const uint32_t* offsets,
    uint64_t* gaussian_keys_unsorted,
    uint32_t* gaussian_values_unsorted,
    int* radii,
    dim3 grid) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= P)
        return;

    // Generate no key/value pair for invisible Gaussians
    if (radii[idx] > 0) {
        // Find this Gaussian's offset in buffer for writing keys/values.
        uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
        uint2 rect_min, rect_max;

        getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

        // For each tile that the bounding rect overlaps, emit a
        // key/value pair. The key is |  tile ID  |      depth      |,
        // and the value is the ID of the Gaussian. Sorting the values
        // with this key yields Gaussian IDs in a list, such that they
        // are first sorted by tile and then by depth.
        for (int y = rect_min.y; y < rect_max.y; y++) {
            for (int x = rect_min.x; x < rect_max.x; x++) {
                uint64_t key = y * grid.x + x;
                key <<= 32;
                key |= *((uint32_t*)&depths[idx]);
                gaussian_keys_unsorted[off] = key;
                gaussian_values_unsorted[off] = idx;
                off++;
            }
        }
    }
}

// Check keys to see if it is at the start/end of one tile's range in
// the full sorted list. If yes, write start/end of this tile.
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges) {
    auto idx = cg::this_grid().thread_rank();
    if (idx >= L)
        return;

    // Read tile ID from key. Update start/end of tile range if at limit.
    uint64_t key = point_list_keys[idx];
    uint32_t currtile = key >> 32;
    if (idx == 0)
        ranges[currtile].x = 0;
    else {
        uint32_t prevtile = point_list_keys[idx - 1] >> 32;
        if (currtile != prevtile) {
            ranges[prevtile].y = idx;
            ranges[currtile].x = idx;
        }
    }
    if (idx == L - 1)
        ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
    int P,
    float* means3D,
    float* viewmatrix,
    float* projmatrix,
    bool* present) {
    checkFrustum<<<(P + 255) / 256, 256>>>(
        P,
        means3D,
        viewmatrix, projmatrix,
        present);
}

/**
 * @brief 从给定的内存块中恢复GeometryState对象
 * 
 * @param chunk 
 * @param P 所有高斯球的数量
 * @return CudaRasterizer::GeometryState 
 */
CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P) {
    GeometryState geom;//将用于存储从内存块中提取的数据
    obtain(chunk, geom.depths, P, 128);//从内存块中提取指定数量的数据，并根据给定的对齐方式调整指针的位置。
    obtain(chunk, geom.clamped, P * 3, 128);
    obtain(chunk, geom.internal_radii, P, 128);
    obtain(chunk, geom.means2D, P, 128);
    obtain(chunk, geom.cov3D, P * 6, 128);
    obtain(chunk, geom.conic_opacity, P, 128);
    obtain(chunk, geom.rgb, P * 3, 128);
    obtain(chunk, geom.tiles_touched, P, 128);
    hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);//使用CUB库的扫描操作计算tiles_touched的累积和，并将结果存储在geom.tiles_touched中。
    obtain(chunk, geom.scanning_space, geom.scan_size, 128);
    obtain(chunk, geom.point_offsets, P, 128);
    return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N) {
    ImageState img;
    obtain(chunk, img.accum_alpha, N, 128);
    obtain(chunk, img.n_contrib, N, 128);
    obtain(chunk, img.ranges, N, 128);
    return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P) {
    BinningState binning;
    obtain(chunk, binning.point_list, P, 128);
    obtain(chunk, binning.point_list_unsorted, P, 128);
    obtain(chunk, binning.point_list_keys, P, 128);
    obtain(chunk, binning.point_list_keys_unsorted, P, 128);
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, binning.sorting_size,
        binning.point_list_keys_unsorted, binning.point_list_keys,
        binning.point_list_unsorted, binning.point_list, P);
    obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
    return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
    std::function<char*(size_t)> geometryBuffer,
    std::function<char*(size_t)> binningBuffer,
    std::function<char*(size_t)> imageBuffer,
    const int P, int D, int M,
    const float* background,
    const int width, int height,
    const float* means3D,
    const float* shs,
    const float* colors_precomp,
    const float* opacities,
    const float* scales,
    const float scale_modifier,
    const float* rotations,
    const float* cov3D_precomp,
    const float* viewmatrix,
    const float* projmatrix,
    const float* cam_pos,
    const float tan_fovx, float tan_fovy,
    const bool prefiltered,
    float* out_color,
    int* radii,
    bool debug) 
{
    const float focal_y = static_cast<float>(height) / (2.f * tan_fovy);//获取焦距y
    const float focal_x = static_cast<float>(width) / (2.f * tan_fovx);//获取焦距x

    // 动态调整缓冲区大小并初始化内存空间，在preprocess的时候才进行初始化吧~
    size_t chunk_size = required<GeometryState>(P);//P为高斯球的数量
    char* chunkptr = geometryBuffer(chunk_size);//获取缓冲区
    GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

    if (radii == nullptr) {
        radii = geomState.internal_radii;
    }

    dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    dim3 block(BLOCK_X, BLOCK_Y, 1);

    // Dynamically resize image-based auxiliary buffers during training
    size_t img_chunk_size = required<ImageState>(width * height);
    char* img_chunkptr = imageBuffer(img_chunk_size);
    ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

    if (NUM_CHANNELS != 3 && colors_precomp == nullptr) {//注意需要RGB数据，如果不是RGB数据，需要提供预先计算的高斯颜色
        throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
    }

    // Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)（预处理高斯函数参数）
    CHECK_CUDA(FORWARD::preprocess(
                   P, D, M,
                   means3D,
                   (glm::vec3*)scales,
                   scale_modifier,
                   (glm::vec4*)rotations,
                   opacities,
                   shs,
                   geomState.clamped,
                   cov3D_precomp,
                   colors_precomp,
                   viewmatrix, projmatrix,
                   (glm::vec3*)cam_pos,
                   width, height,
                   focal_x, focal_y,
                   tan_fovx, tan_fovy,
                   radii,//高斯球2投影的半径
                   geomState.means2D,
                   geomState.depths,
                   geomState.cov3D,
                   geomState.rgb,
                   geomState.conic_opacity,
                   tile_grid,
                   geomState.tiles_touched,//2D高斯函数覆盖的矩形范围
                   prefiltered),
               debug)

    // Compute prefix sum over full list of touched tile counts by Gaussians
    // E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
    CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)
    //根据上面获取的geomState.tiles_touched来计算的
    // 用于对被高斯函数触碰到的瓦片数量列表进行前缀和计算。前缀和是将列表中每个位置的值替换为该位置之前所有值的总和的一种操作。这里的目的是统计出每个高斯函数触碰到的瓦片总数，以便后续的渲染操作。
    // 调用这个函数后，geomState.tiles_touched 数组中存储的值就是前缀和的结果，表示每个高斯函数触碰到的瓦片总数。

    // Retrieve total number of Gaussian instances to launch and resize aux buffers
    // 这段代码用于获取要启动的高斯实例的总数，并调整辅助缓冲区的大小以适应这些实例的数量
    int num_rendered;
    CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

    size_t binning_chunk_size = required<BinningState>(num_rendered);
    char* binning_chunkptr = binningBuffer(binning_chunk_size);
    BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

    // For each instance to be rendered, produce adequate [ tile | depth ] key
    // and corresponding dublicated Gaussian indices to be sorted
    // 每个要渲染的高斯实例生成适当的 [tile | depth] 键，以及相应的重复的高斯索引，以便进行排序。
    duplicateWithKeys<<<(P + 255) / 256, 256>>>(
        P,
        geomState.means2D,
        geomState.depths,
        geomState.point_offsets,
        binningState.point_list_keys_unsorted,
        binningState.point_list_unsorted,
        radii,
        tile_grid)
        CHECK_CUDA(, debug)

            int bit = getHigherMsb(tile_grid.x * tile_grid.y);

    // Sort complete list of (duplicated) Gaussian indices by keys
    CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
                   binningState.list_sorting_space,
                   binningState.sorting_size,
                   binningState.point_list_keys_unsorted, binningState.point_list_keys,
                   binningState.point_list_unsorted, binningState.point_list,
                   num_rendered, 0, 32 + bit),
               debug)

    CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

    // Identify start and end of per-tile workloads in sorted list
    if (num_rendered > 0)
        identifyTileRanges<<<(num_rendered + 255) / 256, 256>>>(
            num_rendered,
            binningState.point_list_keys,
            imgState.ranges);
    CHECK_CUDA(, debug)

    // Let each tile blend its range of Gaussians independently in parallel
    const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
    CHECK_CUDA(FORWARD::render(
                   tile_grid, block,
                   imgState.ranges,
                   binningState.point_list,
                   width, height,
                   geomState.means2D,
                   feature_ptr,
                   geomState.conic_opacity,
                   imgState.accum_alpha,
                   imgState.n_contrib,
                   background,
                   out_color),
               debug)

    return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
    const int P, int D, int M, int R,
    const float* background,
    const int width, int height,
    const float* means3D,
    const float* shs,
    const float* colors_precomp,
    const float* scales,
    const float scale_modifier,
    const float* rotations,
    const float* cov3D_precomp,
    const float* viewmatrix,
    const float* projmatrix,
    const float* campos,
    const float tan_fovx, float tan_fovy,
    const int* radii,
    char* geom_buffer,
    char* binning_buffer,
    char* img_buffer,
    const float* dL_dpix,
    float* dL_dmean2D,
    float* dL_dconic,
    float* dL_dopacity,
    float* dL_dcolor,
    float* dL_dmean3D,
    float* dL_dcov3D,
    float* dL_dsh,
    float* dL_dscale,
    float* dL_drot,
    bool debug) {
    GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
    BinningState binningState = BinningState::fromChunk(binning_buffer, R);
    ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

    if (radii == nullptr) {
        radii = geomState.internal_radii;
    }

    const float focal_y = height / (2.0f * tan_fovy);
    const float focal_x = width / (2.0f * tan_fovx);

    const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    const dim3 block(BLOCK_X, BLOCK_Y, 1);

    // Compute loss gradients w.r.t. 2D mean position, conic matrix,
    // opacity and RGB of Gaussians from per-pixel loss gradients.
    // If we were given precomputed colors and not SHs, use them.
    const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
    CHECK_CUDA(BACKWARD::render(
                   tile_grid,
                   block,
                   imgState.ranges,
                   binningState.point_list,
                   width, height,
                   background,
                   geomState.means2D,
                   geomState.conic_opacity,
                   color_ptr,
                   imgState.accum_alpha,
                   imgState.n_contrib,
                   dL_dpix,
                   (float3*)dL_dmean2D,
                   (float4*)dL_dconic,
                   dL_dopacity,
                   dL_dcolor),
               debug)

    // Take care of the rest of preprocessing. Was the precomputed covariance
    // given to us or a scales/rot pair? If precomputed, pass that. If not,
    // use the one we computed ourselves.
    const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
    CHECK_CUDA(BACKWARD::preprocess(P, D, M,
                                    (float3*)means3D,
                                    radii,
                                    shs,
                                    geomState.clamped,
                                    (glm::vec3*)scales,
                                    (glm::vec4*)rotations,
                                    scale_modifier,
                                    cov3D_ptr,
                                    viewmatrix,
                                    projmatrix,
                                    focal_x, focal_y,
                                    tan_fovx, tan_fovy,
                                    (glm::vec3*)campos,
                                    (float3*)dL_dmean2D,
                                    dL_dconic,
                                    (glm::vec3*)dL_dmean3D,
                                    dL_dcolor,
                                    dL_dcov3D,
                                    dL_dsh,
                                    (glm::vec3*)dL_dscale,
                                    (glm::vec4*)dL_drot),
               debug)
}